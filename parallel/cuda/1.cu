#include "hip/hip_runtime.h"
/*----------------------------------------------------------------
 *
 * Actividad de programación: Solución para Lisp++
 * Fecha: 26-Nov-2017
 * Autor: A00225851 Iván Alejandro Soto Velázquez
 *
 *--------------------------------------------------------------*/


#include <bits/stdc++.h>
#include "utils/cppheader.h"

using namespace std;

const int B = 123;
const int N = B * 111; /* 123 * 111 */

int T;
int n;
char s[N];
struct thread {
  int balance;
  int b, e;
} threads[111];

int find_error_index(char *s, int b, int e, int balance) {
  for (int i = b; i < e; i++) {
    if (s[i]== '(') {
      balance++;
    } else if (s[i] == ')') {
      balance--;
      if (balance < 0) return i - b;
    }
  }
  assert(false);
  return -1;
}

int solve(char *s, thread **threads) {
  int prefix_sum = 0;
  int len = 0;
  for (int i = 0; i < T; i++) {
    if (prefix_sum + threads[i]->balance < 0) {
      len += find_error_index(s, threads[i]->b, threads[i]->e, prefix_sum);
      return len;
    }
    len += threads[i]->e - threads[i]->b;
    prefix_sum += threads[i]->balance;
  }
  if (prefix_sum == 0) return -1;
  assert(len == n);
  return len;
}

__global__ void kernel_solve(char *s, int *range, int *n, thread **threads) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int b = i * (*range);
  int e = min(*n, (i + 1) * (*range));
  int balance = 0;
  threads[i]->b = b;
  threads[i]->e = e;
  for (int i = b; i < e; i++) {
    if (s[i] == '(') {
      balance++;
    } else if (s[i] == ')') {
      balance--;
    } else assert(0);
  }
  threads[i]->balance = balance;
}

int main() {
  scanf("%s", s);
  double ms = 0;
  Timer t;
  t.start();
  int *n = (int *) malloc(sizeof(int));
  int *d_n;
  hipMalloc((void **) &d_n, sizeof(int));
  char *d_s;
  hipMalloc((void **) &d_s, sizeof(s));
  *n = strlen(s);
  hipMemcpy(d_n, n, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_s, s, sizeof(s), hipMemcpyHostToDevice);
  T = min(100, *n);
  int *range = (int *) malloc(sizeof(int));
  *range = (*n + T - 1) / T;
  int *d_range;
  hipMalloc((void **) &d_range, sizeof(int));
  hipMemcpy(d_range, range, sizeof(int), hipMemcpyHostToDevice);
  thread *threads = (thread *) malloc(sizeof(thread) * T);
  for (int i = 0; i < T; i++) {
    threads[i] = (thread *) malloc(sizeof(thread));
  }
  thread *d_threads;
  hipMalloc((void **) &d_threads, sizeof(thread) * T);
  for (int i = 0; i < T; i++) {
    hipMalloc((void **) &d_threads[i], sizeof(thread));
    hipMemcpy(d_threads[i], threads[i], sizeof(thread), hipMemcpyHostToDevice);
  }
  kernel_solve<<<(N + T - 1) / T, T>>>(d_s, d_range, d_n, d_threads);
  for (int i = 0; i < T; i++) {
    hipMemcpy(threads[i], d_threads[i], sizeof(thread), hipMemcpyDeviceToHost);
  }
  int ans = solve(s, threads);
  ms += t.stop();
  printf("%d\n", ans);
  fprintf(stderr, "Program's execution time: %.6lf ms\n", ms);
  return 0;
}
